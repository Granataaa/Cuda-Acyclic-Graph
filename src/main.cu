#include <iostream>
#include <hip/hip_runtime.h>
#include "graph_utils.h"
#include "scan.h"

int main() {
    int n;
    int* adj = loadGraphFromFile("data/sample_graph10.txt", &n);
    check_acyclic(adj, n);
    delete[] adj;
    return 0;
}